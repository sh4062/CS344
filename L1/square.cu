#include "hip/hip_runtime.h"
#include<stdio.h>


__global__ void square(float *d_out,float * d_in){
    int idx = nv.x;
    float f = d_in[idx];
    d_out[idx]= f*f;
}
int main(int argc,char** argv){
    printf("CS344\n");
    const int ARRAY_SIZE = 10;
    const int ARRAY_BYTES = ARRAY_SIZE*sizeof(float);

    //generate the input array on the host
    float h_in[ARRAY_SIZE];
    for(int i = 0;i<ARRAY_SIZE;i++){
        h_in[i]=(float)(i);
    }
    float h_out[ARRAY_SIZE];

    //declare GPU memory pointers
    float * d_in;
    float * d_out;
    //allocate GPU memoryy
    hipMalloc((void**)&d_in,ARRAY_BYTES);
    hipMalloc((void**)&d_out,ARRAY_BYTES);
    //transfer to GPU
    printf("transfer to GPU\n");
    for(int i = 0;i<ARRAY_SIZE;i++){
        printf("%f",h_in[i]);
        printf(((i%4)!=3)?"\t":"\n");
    }
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
    //launch kernal
    square<<<1,ARRAY_SIZE>>>(d_out,d_in);
    //copy back
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
    //print out results
    for(int i = 0;i<ARRAY_SIZE;i++){
        printf("%f",h_out[i]);
        printf(((i%4)!=3)?"\t":"\n");
    }
    //free cudamem
    hipFree(d_in);
    hipFree(d_out);
    return 0;
    
}
